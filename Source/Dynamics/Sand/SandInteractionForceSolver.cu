#include "hip/hip_runtime.h"
#include "Dynamics/Sand/SandInteractionForceSolver.h"

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <cmath>

#include "Core/Utility/CTimer.h"
#include "Core/Utility/CudaRand.h"

namespace PhysIKA {

void SandInteractionForceSolver::addSDF(DistanceField3D<DataType3f>& sdf, int rigidid)
{

    float* tmpa = new float[10];
    memset(tmpa, 0, sizeof(float) * 10);
    DeviceArray<double> devTmpa;
    devTmpa.resize(10);
    hipMemcpy(devTmpa.begin(), tmpa, sizeof(float) * 10, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    delete[] tmpa;
    devTmpa.release();

    if (rigidid < 0)
        rigidid = m_sdfMap.size();
    m_sdfMap[rigidid] = sdf;
}

__global__ void SandIFS_updateSinkInfo(
    DeviceDArray<double>             topH,
    DeviceDArray<double>             botH,
    DeviceDArray<Vector3d>           topNormal,
    DeviceDArray<Vector3d>           botNormal,
    DeviceDArray<Vector3d>           positions,
    DeviceHeightField1d              land,
    DistanceField3D<DataType3f>      sdf,
    DeviceArray<PBDBodyInfo<double>> body,
    int                              sdfid,
    double                           dh)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= positions.size())
        return;

    PBDBodyInfo<double>& bodyi = body[sdfid];

    Vector3d pointd = positions[tid];
    Vector3d normaltop;
    Vector3d normalbot;

    double hland = land.get(pointd[0], pointd[2]);
    double curh  = pointd[1] /*+ hland*/;
    double htop  = hland;
    double hbot  = curh;
    while (curh > hland)
    {
        pointd    = positions[tid];
        pointd[1] = curh;
        bodyi.pose.invTransform(pointd);
        Vector3f pointf(pointd[0], pointd[1], pointd[2]);

        Vector3f normalf;
        float    dis;
        sdf.getDistance(pointf, dis, normalf);
        normalf *= -1;

        if (dis <= 0)
        {

            //Vector3d tmpp = positions[tid];
            //tmpp[1] = curh;
            //tmpp -= bodyi.pose.position;
            //double tmpnorm = tmpp.norm();
            //printf("detected: %lf,  %lf; %lf %lf %lf;  %lf %lf %lf\n", dis, tmpnorm,
            //	bodyi.pose.position[0], bodyi.pose.position[1], bodyi.pose.position[2],
            //	positions[tid][0], curh, positions[tid][2]
            //	);

            if (htop < curh)
            {
                htop      = curh;
                normaltop = Vector3d(normalf[0], normalf[1], normalf[2]);
                bodyi.pose.rotate(normaltop);
            }
            if (hbot > curh)
            {
                hbot      = curh;
                normalbot = Vector3d(normalf[0], normalf[1], normalf[2]);
                bodyi.pose.rotate(normalbot);
            }
        }

        curh -= dh;
    }

    topH[tid]      = htop;
    botH[tid]      = hbot;
    topNormal[tid] = normaltop;
    botNormal[tid] = normalbot;
}

__global__ void SandIFS_computeBuoyancy(
    DeviceDArray<Vector3d>            buoF,
    DeviceDArray<Vector3d>            buoT,
    DeviceDArray<double>              relvDf,
    DeviceDArray<double>              topH,
    DeviceDArray<double>              botH,
    DeviceDArray<Vector3d>            botNor,
    DeviceDArray<Vector3d>            posArr,
    DeviceDArray<Vector3d>            parVel,
    DeviceDArray<double>              massArr,
    DeviceHeightField1d               land,
    DeviceDArray<PBDBodyInfo<double>> body,
    int                               bodyid,
    double                            gravity)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= massArr.size())
        return;

    double   htop   = topH[tid];
    double   hbot   = botH[tid];
    Vector3d pointd = posArr[tid];
    if (htop > hbot)
    {
        int                  botid = tid * 2;
        PBDBodyInfo<double>& bodyi = body[bodyid];

        double hland = land.get(pointd[0], pointd[2]);
        double force = gravity * massArr[tid] * (htop - hbot) / (pointd[1] - hland);

        buoF[botid][0] = 0;  // force * botNor[tid][0] / (botNor[tid][1] - 1e-2);
        buoF[botid][1] = force;
        buoF[botid][2] = 0;  // force * botNor[tid][2] / (botNor[tid][1] - 1e-2);

        //Vector3d force(0, buoF[botid], 0);
        pointd[1]   = hbot;
        buoT[botid] = (pointd - bodyi.pose.position).cross(buoF[botid]);

        //Vector3d tmpv = buoF[botid];// (pointd - body[bodyid].pose.position);
        //if (buoF[botid][0] != 0 || buoF[botid][1] != 0 || buoF[botid][2] != 0)
        //{
        //	printf("%d: F, %lf;   val, %lf %lf %lf\n", tid, buoF[tid*2][1], htop, hbot, hland);
        //}
        relvDf[botid] = buoF[botid].dot(bodyi.linVelocity - parVel[tid])
                        + buoT[botid].dot(bodyi.angVelocity);
    }
    else
    {
        buoF[tid * 2]   = Vector3d();
        buoT[tid * 2]   = Vector3d();
        relvDf[tid * 2] = 0.0;
    }
    buoF[tid * 2 + 1]   = Vector3d();
    buoT[tid * 2 + 1]   = Vector3d();
    relvDf[tid * 2 + 1] = 0.0;
}

void SandInteractionForceSolver::updateSinkInfo(int i)
{

    m_topH.resize(m_particlePos->size());
    m_botH.resize(m_particlePos->size());
    m_topNormal.resize(m_particlePos->size());
    m_botNormal.resize(m_particlePos->size());

    m_topH.reset();
    m_botH.reset();
    m_topNormal.reset();
    m_botNormal.reset();

    int rid = i;
    if (m_prigids)
        rid = (*m_prigids)[i]->getId();

    cuExecute(m_particlePos->size(), SandIFS_updateSinkInfo, m_topH, m_botH, m_topNormal, m_botNormal, *m_particlePos, *m_land,
              //(*m_sdfs)[i],
              m_sdfMap[rid],
              *m_body,
              i,
              m_sampleSize);

    ////  debug
    //HostDArray<double> hostTop;
    //hostTop.resize(m_topH.size());
    //Function1Pt::copy(hostTop, m_topH);

    //HostDArray<double> hostBot;
    //hostBot.resize(m_botH.size());
    //Function1Pt::copy(hostBot, m_botH);

    //HostDArray<Vector3d> hostpos;
    //hostpos.resize(m_particlePos->size());
    //Function1Pt::copy(hostpos, *m_particlePos);

    //double* hostland = new double[m_land->Nx() * m_land->Ny()];
    //hipMemcpy2D(hostland, m_land->Nx(), m_land->GetDataPtr(), m_land->Pitch(),
    //	m_land->Nx(), m_land->Ny(), hipMemcpyDeviceToHost);

    //hostTop.release();
    //hostBot.release();
    //hostpos.release();
    //delete[] hostland;
}

//void SandInteractionForceSolver::computeBuoyancy()
//{
//	if (!m_body || m_body->size() <= 0)
//		return;

//	for (int i = 0; i < m_body->size(); ++i)
//	{
//		this->computeSingleBuoyance(i);
//	}
//}
void SandInteractionForceSolver::computeSingleBuoyance(int i, Real dt)
{
    if (!m_particlePos || m_particlePos->size() <= 0)
        return;

    m_buoyancyF.resize(m_particlePos->size() * 2);
    m_buoyancyT.resize(m_particlePos->size() * 2);
    m_relvDf.resize(m_particlePos->size() * 2);
    m_buoyancyF.reset();
    m_buoyancyT.reset();
    m_relvDf.reset();

    cuExecute(m_particlePos->size(), SandIFS_computeBuoyancy, m_buoyancyF, m_buoyancyT, m_relvDf, m_topH, m_botH, m_botNormal, *m_particlePos, *m_particleVel, *m_particleMass, *m_land,
              //*m_body,
              m_averageBodyInfo,
              i,
              m_gravity);

    //m_devArr1d.resize(m_buoyancyF.size());
    //Function1Pt::copy(m_devArr1d, m_buoyancyF);
    //m_devArr3d.resize(m_buoyancyT.size());
    //Function1Pt::copy(m_devArr3d, m_buoyancyT);

    //double buoF = 1;
    Vector3d buoF = thrust::reduce(thrust::device, m_buoyancyF.begin(), m_buoyancyF.begin() + m_buoyancyF.size(), Vector3d(), thrust::plus<Vector3d>());

    Vector3d buoT /*= thrust::reduce(thrust::device, m_buoyancyT.begin(), m_buoyancyT.begin() + m_buoyancyT.size(),
			Vector3d(), thrust::plus<Vector3d>())*/
        (0, 0, 0);

    double relvdf = thrust::reduce(thrust::device, m_relvDf.begin(), m_relvDf.begin() + m_relvDf.size(), ( double )0.0, thrust::plus<double>());

    //// debug
    //HostDArray<Vector3d> hostF;
    //hostF.resize(m_buoyancyF.size());
    //Function1Pt::copy(hostF, m_buoyancyF);

    //HostDArray<Vector3d> hostT;
    //hostT.resize(m_buoyancyT.size());
    //Function1Pt::copy(hostT, m_buoyancyT);

    //HostDArray<double> hostMass;
    //hostMass.resize(m_particleMass->size());
    //Function1Pt::copy(hostMass, *m_particleMass);

    //

    //double tmpv = buoF[1];
    //hostF.release();
    //hostT.release();
    //hostMass.release();

    if ((m_hostBody)[i].invMass > 0)
    {
        m_Abuo = this->_enlargerBuoyancy(buoF[1], buoT, 1.0 / (m_hostBody)[i].invMass);
        m_Abuo = m_Abuo > 1e9 ? 1e9 : m_Abuo;

        if (abs(buoF[1]) > EPSILON)
        {
            m_Abuo /= abs(buoF[1]);
        }
        m_Abuo *= m_gravity;
    }
    else
    {
        m_Abuo = 0.0;
    }
    buoF *= m_Abuo;
    buoT *= m_Abuo;
    relvdf *= m_Abuo;

    double alpha = this->_minEng(buoF, buoT, relvdf, i, dt);
    //double alpha = 1.0;
    buoF *= alpha;
    buoT *= alpha;

    // debug
    Vector3d& debLinv = (m_hostBody + i)->linVelocity;
    Vector3d& debAngv = (m_hostBody + i)->angVelocity;

    if (false)
    {
        printf("**** %d \n", i);

        printf("  Body vel(before BUO):  %lf %lf %lf,  %lf %lf %lf\n",
               debLinv[0],
               debLinv[1],
               debLinv[2],
               debAngv[0],
               debAngv[1],
               debAngv[2]);

        printf("Buoy F: %lf %lf %lf, Buoy T: %lf %lf %lf, Abuo: %lf , Alpha: %lf\n",
               buoF[0],
               buoF[1],
               buoF[2],
               buoT[0],
               buoT[1],
               buoT[2],
               m_Abuo,
               alpha);
    }

    (m_hostBody + i)->linVelocity = m_prevBody[i].linVelocity;
    (m_hostBody + i)->angVelocity = m_prevBody[i].angVelocity;

    this->_applyForceTorque(buoF, buoT, i, dt);

    if (false)
    {
        printf("  Body vel(after BUO):  %lf %lf %lf,  %lf %lf %lf\n",
               debLinv[0],
               debLinv[1],
               debLinv[2],
               debAngv[0],
               debAngv[1],
               debAngv[2]);

        auto& prevPose  = m_prevBody[i].prevPose;
        auto  pbodyPose = (m_hostBody + i)->pose;
        printf("  Body Pos(prev):  %lf %lf %lf,  %lf %lf %lf %lf\n",
               prevPose.position[0],
               prevPose.position[1],
               prevPose.position[2],
               prevPose.rotation[0],
               prevPose.rotation[1],
               prevPose.rotation[2],
               prevPose.rotation[3]);
        printf("  Body Pos(cur) :  %lf %lf %lf,  %lf %lf %lf %lf\n",
               pbodyPose.position[0],
               pbodyPose.position[1],
               pbodyPose.position[2],
               pbodyPose.rotation[0],
               pbodyPose.rotation[1],
               pbodyPose.rotation[2],
               pbodyPose.rotation[3]);
    }
}

__global__ void SandIFS_DragForceVel(
    DeviceDArray<Vector3d>           dragF,
    DeviceDArray<Vector3d>           dragT,
    DeviceDArray<double>             relvDf,
    DeviceDArray<double>             topH,
    DeviceDArray<double>             botH,
    DeviceDArray<Vector3d>           topN,
    DeviceDArray<Vector3d>           botN,
    DeviceDArray<double>             massArr,
    DeviceDArray<Vector3d>           posArr,
    DeviceDArray<Vector3d>           velArr,
    DeviceHeightField1d              land,
    DeviceArray<PBDBodyInfo<double>> body,
    int                              bodyid,
    double                           rho,
    double                           mu,
    double                           gravity,
    double                           alpha,
    double                           beta,
    double                           Cdrag)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= massArr.size())
        return;

    if (topH[tid] <= botH[tid])
        return;

    Vector3d posi  = posArr[tid];
    double   hland = land.get(posi[0], posi[2]);

    // Effective area.
    double effA = massArr[tid] / ((posArr[tid][1] - hland) * rho);

    if (posi[1] > topH[tid])
    {
        double depth = posi[1] - topH[tid];
        int    topid = tid * 2 + 1;
        // Relative velocity.
        Vector3d relp = posi;
        relp[1]       = topH[tid];
        relp -= body[bodyid].pose.position;
        Vector3d relv = body[bodyid].linVelocity + body[bodyid].angVelocity.cross(relp);
        relv -= velArr[tid];

        double effAi = topN[tid].dot(relv) * effA / (abs(topN[tid][1]) + 0.05);

        // Force
        double f = beta * rho * effAi;
        f        = f > 0 ? -f : 0;

        // Frictional drag.
        double relvNorm = relv.norm();
        if (relvNorm > EPSILON)
            effAi /= (relvNorm * relvNorm);
        else
            effAi = 0;
        double f_fric = alpha * rho * gravity * mu * depth * effAi;
        f_fric        = f_fric > 0 ? -f_fric : 0;
        f += f_fric;
        f *= Cdrag;

        dragF[topid] = relv * f;
        dragT[topid] = relp.cross(relv * f);

        relvDf[topid] = dragF[topid].dot(body[bodyid].linVelocity - velArr[tid])
                        + dragT[topid].dot(body[bodyid].angVelocity);
    }
    else
    {
        dragF[tid * 2 + 1]  = Vector3d();
        dragT[tid * 2 + 1]  = Vector3d();
        relvDf[tid * 2 + 1] = 0.0;
    }

    // bottom contact.
    {
        double depth = posi[1] - botH[tid];
        int    botid = 2 * tid;

        // Relative velocity.
        Vector3d relp = posi;
        relp[1]       = botH[tid];
        relp -= body[bodyid].pose.position;
        Vector3d relv = body[bodyid].linVelocity + body[bodyid].angVelocity.cross(relp);
        relv -= velArr[tid];

        double effAi = botN[tid].dot(relv) * effA / (abs(botN[tid][1]) + 0.05);

        // Force
        double f = beta * rho * effAi;
        f        = f > 0 ? -f : 0;

        // Frictional drag.
        double relvNorm = relv.norm();
        if (relvNorm > EPSILON)
            effAi /= (relvNorm * relvNorm);
        else
            effAi = 0;

        double f_fric = alpha * rho * gravity * mu * depth * effAi;

        //if (body[bodyid].linVelocity[0] != 0 || body[bodyid].linVelocity[1] != 0 || body[bodyid].linVelocity[2] != 0)
        //	printf("Vel>0 : %lf  %lf %lf, Norm: %lf %lf %lf\n", f_fric, botN[tid].dot(relv), effAi,
        //		relv[0], relv[1], relv[2]);

        f_fric = f_fric > 0 ? -f_fric : 0;
        f += f_fric;
        f *= Cdrag;

        dragF[botid] = relv * f;
        dragT[botid] = relp.cross(relv * f);

        relvDf[botid] = dragF[botid].dot(body[bodyid].linVelocity - velArr[tid])
                        + dragT[botid].dot(body[bodyid].angVelocity);
    }
}

void SandInteractionForceSolver::computeSingleDragForce(int i, Real dt)
{
    m_dragF.resize(m_particlePos->size() * 2);
    m_dragT.resize(m_particlePos->size() * 2);
    m_relvDf.resize(m_particlePos->size() * 2);

    m_dragF.reset();
    m_dragT.reset();
    m_relvDf.reset();

    // debug
    CTimer timer;
    timer.start();

    cuExecute(m_particlePos->size(), SandIFS_DragForceVel, m_dragF, m_dragT, m_relvDf, m_topH, m_botH, m_topNormal, m_botNormal, *m_particleMass, *m_particlePos, *m_particleVel, *m_land, *m_body,
              //m_averageBodyInfo,
              i,
              m_rho,
              (m_sandMu + m_hostBody[i].mu) / 2.0,
              m_gravity,
              m_alpha,
              m_beta,
              m_Cdrag);

    timer.stop();
    //printf("      DragForce kernel time:  %lf \n", timer.getElapsedTime());

    //// debug
    //HostDArray<Vector3d> hostF;
    //hostF.resize(m_dragF.size());
    //Function1Pt::copy(hostF, m_dragF);

    //HostDArray<Vector3d> hostT;
    //hostT.resize(m_dragT.size());
    //Function1Pt::copy(hostT, m_dragT);

    //hostF.release();
    //hostT.release();

    timer.start();

    Vector3d dragF = thrust::reduce(thrust::device, m_dragF.begin(), m_dragF.begin() + m_dragF.size(), Vector3d(), thrust::plus<Vector3d>());

    Vector3d dragT = thrust::reduce(thrust::device, m_dragT.begin(), m_dragT.begin() + m_dragT.size(), Vector3d(), thrust::plus<Vector3d>());

    double relvdf = thrust::reduce(thrust::device, m_relvDf.begin(), m_relvDf.begin() + m_relvDf.size(), ( double )0.0, thrust::plus<double>());

    timer.stop();
    //printf("      DragForce summation time:  %lf \n", timer.getElapsedTime());

    // debug

    Vector3d& debLinv = (m_hostBody + i)->linVelocity;
    Vector3d& debAngv = (m_hostBody + i)->angVelocity;
    if (false)
    {
        printf("  Body vel (before DRAG):  %lf %lf %lf,  %lf %lf %lf\n",
               debLinv[0],
               debLinv[1],
               debLinv[2],
               debAngv[0],
               debAngv[1],
               debAngv[2]);
        printf("BEF:   Drag F: %lf %lf %lf, Drag T: %lf %lf %lf, Reldf: %lf \n",
               dragF[0],
               dragF[1],
               dragF[2],
               dragT[0],
               dragT[1],
               dragT[2],
               relvdf);
    }

    //if (dragF.norm() > 0 || dragT.norm() > 0)
    this->_stableDamping(i, dragF, dragT, dt);

    double alpha = this->_minEng(dragF, dragT, relvdf, i, dt);
    //double alpha = 1.0;
    dragF *= alpha;
    dragT *= alpha;

    if (false)
    {
        printf("AFT:   Drag F: %lf %lf %lf, Drag T: %lf %lf %lf, Alpha: %lf \n",
               dragF[0],
               dragF[1],
               dragF[2],
               dragT[0],
               dragT[1],
               dragT[2],
               alpha);
    }

    this->_applyForceTorque(dragF, dragT, i, dt);

    if (false)
    {
        printf("  Body vel(after DRAG):  %lf %lf %lf,  %lf %lf %lf\n",
               debLinv[0],
               debLinv[1],
               debLinv[2],
               debAngv[0],
               debAngv[1],
               debAngv[2]);
    }
}

__global__ void SandIFS_updateParticleVel(
    DeviceDArray<Vector3d>           dVel,
    DeviceDArray<Vector3d>           parVel,
    DeviceDArray<double>             massArr,
    DeviceDArray<Vector3d>           posArr,
    DeviceDArray<double>             topH,
    DeviceDArray<double>             botH,
    DeviceDArray<Vector3d>           topN,
    DeviceDArray<Vector3d>           botN,
    DeviceHeightField1d              land,
    DeviceArray<PBDBodyInfo<double>> body,
    int                              bodyid,
    double                           sampleDl,
    double                           rho_s, /*double rho_r,*/
    double                           e,
    double                           Chorizon,
    double                           Cvertical,
    double                           Cprob = 10000)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= massArr.size())
        return;

    if (topH[tid] <= botH[tid])
        return;

    Vector3d posi  = posArr[tid];
    double   hsand = posi[1] - land.get(posi[0], posi[2]);

    if (hsand < EPSILON)
        return;

    Vector3d dvel;
    //double curh = topH[tid];
    //while (curh > botH[tid])
    //{
    //	Vector3d curp = posi;
    //	curp[1] = curh;
    //	Vector3d velObj = body[bodyid].getVelocityAt(curp);
    //	dvel += (velObj - parVel[tid])*((1.0 + e) * sampleDl);

    //	curh -= sampleDl;
    //}

    //dvel /= (topH[tid] - botH[tid]);

    posi[1]         = botH[tid];
    Vector3d velObj = body[bodyid].getVelocityAt(posi);
    dvel            = (velObj - parVel[tid]) * (1.0 + e);
    double dvelN    = (dvel.dot(botN[tid]));
    if (dvelN > 0)
    {
        dvel = botN[tid] * dvelN;
        //dvel = topN[tid] * (dvel.dot(topN[tid]));
    }

    dvel[0] *= Chorizon;
    dvel[2] *= Chorizon;
    dvel[1] *= Cvertical;

    double     prob = (topH[tid] - botH[tid]) / hsand * Cprob;
    RandNumber gen(posi[0] * 7000 + posi[2] * 999999);
    double     probval = gen.Generate();

    if (probval >= prob)
    {
        dvel = Vector3d();
    }

    dvel[1]   = dvel[1] > 0.0 ? 0.0 : dvel[1];
    dVel[tid] = dvel;

    ////dVel[tid] = Vector3d();
    //Vector3d finalvel = parVel[tid];// +dVel[tid];
    //Vector3d objLinv = body[bodyid].linVelocity;
    //printf("Vel final: %lf %lf %lf;  %lf %lf %lf\n", finalvel[0], finalvel[1], finalvel[2],
    //	objLinv[0], objLinv[1], objLinv[2]);

    //parVel[tid][1] = 0.0;
}

__global__ void SandIFS_updateParticleVel_Stick(
    DeviceDArray<Vector3d>           dVel,
    DeviceDArray<Vector3d>           parVel,
    DeviceDArray<double>             massArr,
    DeviceDArray<Vector3d>           posArr,
    DeviceDArray<double>             topH,
    DeviceDArray<double>             botH,
    DeviceDArray<Vector3d>           topN,
    DeviceDArray<Vector3d>           botN,
    DeviceHeightField1d              land,
    DeviceArray<PBDBodyInfo<double>> body,
    int                              bodyid,
    double                           sampleDl,
    double                           rho_s,
    /*double rho_r,*/ double         e,
    double                           csHorizon,
    double                           csVertial)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= massArr.size())
        return;

    if (topH[tid] <= botH[tid])
        return;

    Vector3d posi  = posArr[tid];
    double   hsand = posi[1] - land.get(posi[0], posi[2]);

    if (hsand < EPSILON)
        return;

    Vector3d dvel;
    double   curh = topH[tid];
    //while (curh > botH[tid])
    //{
    //	Vector3d curp = posi;
    //	curp[1] = curh;
    //	Vector3d velObj = body[bodyid].getVelocityAt(curp);
    //	dvel += (velObj - parVel[tid])*((1.0 + e) * sampleDl);

    //	curh -= sampleDl;
    //}
    //dvel *= (body[bodyid].rho /
    //	(massArr[tid] + massArr[tid] / rho_s * body[bodyid].rho * (topH[tid] - botH[tid])));

    while (curh > botH[tid])
    {
        Vector3d curp   = posi;
        curp[1]         = curh;
        Vector3d velObj = /*body[bodyid].linVelocity;*/ body[bodyid].getVelocityAt(curp);
        dvel += velObj * sampleDl;

        curh -= sampleDl;

        //// debug
        //if (abs(velObj[0]) >EPSILON || abs(velObj[1]) > EPSILON || abs(velObj[2]) > EPSILON)
        //{
        //	printf("velObj:  %lf %lf %lf\n", velObj[0], velObj[1], velObj[2]);
        //}
    }
    dvel /= hsand;  // (topH[tid] - botH[tid]);

    //// debug
    //if (dvel[0] != 0 || dvel[1] != 0 || dvel[2] != 0)
    //{
    //	printf("DVEL:  %lf %lf %lf\n", dvel[0], dvel[1], dvel[2]);
    //}

    dvel[0] *= csHorizon;
    dvel[2] *= csHorizon;
    dvel[1] *= csVertial;

    //dvel[0] = 0.0;
    //dvel[2] = 0.0;

    dvel[1]   = dvel[1] > 0.0 ? 0.0 : dvel[1];
    dVel[tid] = dvel;

    //parVel[tid][1] = 0.0;
}

void SandInteractionForceSolver::computeParticleInteractVelocity(int i, Real dt)
{
    if (!m_particleVel)
        return;

    m_dVel.resize(m_particlePos->size());
    m_dVel.reset();

    if (m_useStickParticleVelUpdate)
    {
        cuExecute(m_particlePos->size(), SandIFS_updateParticleVel_Stick, m_dVel, *m_particleVel, *m_particleMass, *m_particlePos, m_topH, m_botH, m_topNormal, m_botNormal, *m_land, *m_body, i, m_sampleSize, m_rho, m_e, m_CsHorizon, m_CsVertical);
    }
    else
    {
        cuExecute(m_particlePos->size(), SandIFS_updateParticleVel, m_dVel, *m_particleVel, *m_particleMass, *m_particlePos, m_topH, m_botH, m_topNormal, m_botNormal, *m_land, *m_body, i, m_sampleSize, m_rho, m_e, m_CsHorizon, m_CsVertical, m_Cprob);
    }
}

__global__ void SandIFS_accumulate(
    DeviceDArray<double>             relvDf,
    DeviceDArray<Vector3d>           dforce,
    DeviceDArray<Vector3d>           dtorque,
    DeviceDArray<double>             buoF,
    DeviceDArray<Vector3d>           buoT,
    DeviceDArray<Vector3d>           dragF,
    DeviceDArray<Vector3d>           dragT,
    DeviceDArray<Vector3d>           parVel,
    double                           buoA,
    DeviceArray<PBDBodyInfo<double>> body,
    int                              bodyid)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= parVel.size())
        return;
    PBDBodyInfo<double>& bodyi = body[bodyid];

    int botid = 2 * tid;
    int topid = 2 * tid + 1;

    // bot.
    dforce[botid]  = dragF[botid] + Vector3d(0.0, buoF[botid], 0.0) * buoA;
    dtorque[botid] = dragT[botid] + buoT[botid] * buoA;
    relvDf[botid]  = dforce[botid].dot(bodyi.linVelocity - parVel[tid])
                    + dtorque[botid].dot(bodyi.angVelocity);

    // top.
    dforce[topid]  = dragF[topid] + Vector3d(0.0, buoF[topid], 0.0) * buoA;
    dtorque[topid] = dragT[topid] + buoT[topid] * buoA;
    relvDf[topid]  = dforce[topid].dot(bodyi.linVelocity - parVel[tid])
                    + dtorque[topid].dot(bodyi.angVelocity);
}

//void SandInteractionForceSolver::computeSingleInteractionForce(int i, Real dt, Vector3d& force, Vector3d& torque)
//{
//	m_relvDf.resize(m_particlePos->size() * 2);
//	m_dForce.resize(m_particlePos->size() * 2);
//	m_dTorque.resize(m_particlePos->size() * 2);

//	m_relvDf.reset();
//	m_dForce.reset();
//	m_dTorque.reset();

//	cuExecute(m_particlePos->size(), SandIFS_accumulate,
//		m_relvDf, m_dForce, m_dTorque,
//		m_buoyancyF, m_buoyancyT,
//		m_dragF, m_dragT,
//		*m_particleVel,
//		m_Abuo,
//		*m_body, i
//	);

//	//// debug
//	//HostDArray<double> hostrelvdf;
//	//hostrelvdf.resize(m_relvDf.size());
//	//Function1Pt::copy(hostrelvdf, m_relvDf);

//	//hostrelvdf.release();

//	double relvdf = thrust::reduce(thrust::device,
//		m_relvDf.begin(), m_relvDf.begin() + m_relvDf.size(), (double)0.0, thrust::plus<double>());

//	force = thrust::reduce(thrust::device,
//		m_dForce.begin(), m_dForce.begin() + m_dForce.size(), Vector3d(), thrust::plus<Vector3d>());

//	torque = thrust::reduce(thrust::device,
//		m_dTorque.begin(), m_dTorque.begin() + m_dTorque.size(), Vector3d(), thrust::plus<Vector3d>());

//	double alpha = this->_minEng(force, torque, relvdf, i, dt);

//	force *= alpha;
//	torque *= alpha;

//}

void SandInteractionForceSolver::compute(Real dt)
{
    if (!m_body || m_body->size() <= 0)
        return;
    if (!m_particlePos || m_particlePos->size() <= 0)
        return;

    for (int i = 0; i < m_body->size(); ++i)
    {

        // Check collision filter.
        if (m_prigids && !collisionValid((*m_prigids)[i]))
            continue;

        this->updateSinkInfo(i);
        this->computeSingleBuoyance(i, dt);
        this->_copyHostBodyToGPU(i);
        this->computeSingleDragForce(i, dt);
        this->_copyHostBodyToGPU(i);

        this->computeParticleInteractVelocity(i, dt);

        this->_smoothVelocityChange();
    }
}

void SandInteractionForceSolver::computeSingleBody(int i, Real dt)//TODO
{

    if (!m_hostBody || !m_body || m_body->size() <= 0)
        return;
    if (!m_particlePos || m_particlePos->size() <= 0)
        return;

    // Check collision filter.
    if (m_prigids && !collisionValid((*m_prigids)[i]))
        return;

    //// debug
    //CTimer timer;

    //timer.start();
    this->updateSinkInfo(i);
    //timer.stop();
    //printf("   Interact, Update SinkInfo time:  %lf\n", timer.getElapsedTime());

    //timer.start();
    this->computeSingleBuoyance(i, dt);
    //timer.stop();
    //printf("   Interact, Buoyance time:  %lf\n", timer.getElapsedTime());

    //timer.start();
    this->_copyHostBodyToGPU(i);
    //timer.stop();
    //printf("   Interact, Update body info time:  %lf\n", timer.getElapsedTime());

    //timer.start();
    this->computeSingleDragForce(i, dt);
    //timer.stop();
    //printf("   Interact, Drag force time:  %lf\n", timer.getElapsedTime());

    //timer.start();
    this->_copyHostBodyToGPU(i);
    //timer.stop();
    //printf("   Interact, Update body info time:  %lf\n", timer.getElapsedTime());

    //timer.start();
    this->computeParticleInteractVelocity(i, dt);
    //timer.stop();
    //printf("   Interact, Particle vel change time:  %lf\n", timer.getElapsedTime());

    //timer.start();
    this->_smoothVelocityChange();
    //timer.stop();
    //printf("   Interact, Smooth vel change time:  %lf\n", timer.getElapsedTime());
}

__global__ void SandIFS_smoothVelocityChange(
    DeviceDArray<Vector3d> parVel,
    DeviceDArray<Vector3d> dVel,
    DeviceDArray<double>   massArr,
    DeviceDArray<Vector3d> posArr,
    NeighborList<int>      neighbors,
    SpikyKernel2D<double>  kern,
    double                 smoothlength)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= parVel.size())
        return;

    Vector3d pos_i  = posArr[tid];
    double   weight = 0.0;
    Vector3d dv;

    int nbSize = neighbors.getNeighborSize(tid);
    for (int ne = 0; ne < nbSize; ne++)
    {
        int    j = neighbors.getElement(tid, ne);
        double r = (pos_i - posArr[j]).norm();
        //if (r < EPSILON) continue;
        double w = kern.Weight(r, smoothlength);
        dv += dVel[j] * w;
        weight += w;
    }
    if (weight > EPSILON /*&& nbSize>5*/)
    {
        dv /= weight;
        parVel[tid] += dv;

        /*if (dv[0] < -0.05)
				printf("%d,  Dv:  %lf %lf %lf,  origin: %lf %lf %lf\n", tid,
					dv[0], dv[1], dv[2], dVel[tid][0], dVel[tid][1], dVel[tid][2]);*/
    }
}

__global__ void SandIFS_directUpdateVelocityChange(
    DeviceDArray<Vector3d> parVel,
    DeviceDArray<Vector3d> dVel)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= parVel.size())
        return;

    parVel[tid] += dVel[tid];

    //// debug
    //if (dVel[tid][0] != 0 || dVel[tid][1] != 0 || dVel[tid][2] != 0)
    //{
    //	printf("ParticleVel: %lf %lf %lf\n", parVel[tid][0], parVel[tid][1], parVel[tid][2]);
    //}
}

void SandInteractionForceSolver::_smoothVelocityChange()
{
    if (m_dVel.size() <= 0)
        return;

    if (m_neighbor.isEmpty())
    {

        //static int callCount = 0;
        //++callCount;

        //HostDArray<Vector3d> hostvel;
        //hostvel.resize(m_dVel.size());
        //Function1Pt::copy(hostvel, m_dVel);

        //for (int i = 0; i < hostvel.size(); ++i)
        //{
        //	if (abs(hostvel[i][1]) > 0.001 || abs(hostvel[i][0]) > 0.00 || abs(hostvel[i][2]) > 0.00)
        //	{
        //		printf("  Grid vel: %d,   %lf %lf %lf \n", i, hostvel[i][0], hostvel[i][1], hostvel[i][2]);
        //	}
        //}

        //if (callCount > 170)
        //	return;
        cuExecute(m_particleVel->size(), SandIFS_directUpdateVelocityChange, *m_particleVel, m_dVel);
    }
    else
    {

        cuExecute(m_particlePos->size(), SandIFS_smoothVelocityChange, *m_particleVel, m_dVel, *m_particleMass, *m_particlePos, m_neighbor.getValue(), m_kernel, m_smoothLength

        );
    }
}

double SandInteractionForceSolver::_enlargerBuoyancy(double f, const Vector3d& t, double mass)
{
    double A = mass * m_gravity / m_buoyancyFactor;
    A        = A * (std::exp(f / A) - 1);
    //f *= A;
    //t *= A;
    return A;
}

double SandInteractionForceSolver::_minEng(const Vector3d& dF, const Vector3d& dT, double relvdf, int i, double dt)
{
    if (!m_hostBody)
        return 0.0;
    auto prigid = m_hostBody + i;
    if (!prigid)
        return 0.0;

    float    invMass    = prigid->invMass;
    Vector3d invInertia = prigid->invInertia;

    //Vector3f dTlocal(dT[0], dT[1], dT[2]);
    Vector3d dTlocal = prigid->pose.rotation.getConjugate().rotate(dT);

    double dfInvmDf = dF.dot(dF) * invMass + dTlocal.dot(dTlocal * invInertia);

    if (abs(dfInvmDf) < EPSILON)
        return 0.0;

    double alpha = -relvdf / (dfInvmDf * dt);

    alpha = alpha > 1.0 ? 1.0 : alpha;
    alpha = alpha < 0.0 ? 0.0 : alpha;

    return alpha;
}

void SandInteractionForceSolver::_applyForceTorque(const Vector3d& F, const Vector3d& T, int i, Real dt)
{
    if (!m_hostBody)
        return;

    auto pbody = m_hostBody + i;
    pbody->integrateForceToVelPos(F, T, dt);
    //pbody->integrateForce(F, T, dt);
}

void SandInteractionForceSolver::_stableDamping(int i, Vector3d& F, Vector3d& T, Real dt)
{
    if (!m_hostBody)
        return;

    auto   pbody    = m_hostBody + i;
    double linvnorm = pbody->linVelocity.norm();

    Vector3d tmpv    = F * (pbody->invMass * dt);
    double   maxlinv = tmpv.norm();
    if (tmpv.dot(pbody->linVelocity) < 0 && linvnorm < /*m_gravity*dt * 0.5*/ maxlinv)
    {
        pbody->linVelocity = Vector3d();
        F                  = Vector3d();
    }
    double angvnorm = pbody->angVelocity.norm();
    tmpv            = (T * pbody->invInertia * dt);
    double maxangv  = tmpv.norm();
    if (tmpv.dot(pbody->angVelocity) < 0 && angvnorm < maxangv)
    {
        pbody->angVelocity = Vector3d();
        T                  = Vector3d();
    }
}

void SandInteractionForceSolver::_copyHostBodyToGPU(int i)
{
    hipMemcpy(m_body->begin() + i, m_hostBody + i, sizeof(PBDBodyInfo<double>), hipMemcpyHostToDevice);
}

bool SandInteractionForceSolver::collisionValid(RigidBody2_ptr prigid)
{
    if (!prigid)
        return false;
    bool collide = (prigid->getCollisionFilterGroup() & m_sandCollisionMask);
    collide      = collide && (prigid->getCollisionFilterMask() & m_sandCollisionGroup);
    return collide;
}

void SandInteractionForceSolver::setPreBodyInfo()
{
    if (!m_hostBody || !m_body)
        return;

    m_prevBody.resize(m_body->size());

    cuSafeCall(hipMemcpy(m_prevBody.begin(), m_body->begin(), sizeof(PBDBodyInfo<double>) * m_body->size(), hipMemcpyDeviceToHost));
}

void SandInteractionForceSolver::updateBodyAverageVel(Real dt)
{
    if (m_prevBody.size() <= 0 || !m_hostBody)
        return;

    for (int i = 0; i < m_prevBody.size(); ++i)
    {
        auto pbody             = m_hostBody + i;
        m_prevBody[i].prevPose = m_prevBody[i].pose;
        m_prevBody[i].pose     = pbody->pose;
        m_prevBody[i].updateVelocity(dt);
    }

    //
    m_averageBodyInfo.resize(m_prevBody.size());
    Function1Pt::copy(m_averageBodyInfo, m_prevBody);

    //cuSafeCall(hipMemcpy(m_body->begin(), m_hostBody, sizeof(PBDBodyInfo<double>)*m_body->size(),
    //	hipMemcpyHostToDevice));
}

}  // namespace PhysIKA
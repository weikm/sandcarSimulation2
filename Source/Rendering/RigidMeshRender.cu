#include "hip/hip_runtime.h"
#pragma once
#include <GL/glew.h>
#include "RigidMeshRender.h"
#include "Framework/Topology/TriangleSet.h"
#include "Core/Vector.h"
#include "Framework/Framework/Node.h"
#include "Core/Utility.h"
#include "OpenGLContext.h"

namespace PhysIKA {
IMPLEMENT_CLASS(RigidMeshRender)

RigidMeshRender::RigidMeshRender(std::shared_ptr<Frame<DataType3f>> frame)
    : VisualModule()
    , m_color(Vector3f(0.2f, 0.3, 0.0f))
    , m_meshFrame(frame)
{
}

__global__ void SetupRigidTriangles(
    DeviceArray<float3>                   originVerts,
    DeviceArray<float3>                   vertices,
    DeviceArray<float3>                   normals,
    DeviceArray<float3>                   colors,
    DeviceArray<TopologyModule::Triangle> triangles,
    float3                                color)
{
    int pId = threadIdx.x + (blockIdx.x * blockDim.x);
    if (pId >= triangles.size())
        return;

    TopologyModule::Triangle tri = triangles[pId];
    float3                   v1  = originVerts[tri[0]];
    float3                   v2  = originVerts[tri[1]];
    float3                   v3  = originVerts[tri[2]];

    vertices[3 * pId + 0] = v1;
    vertices[3 * pId + 1] = v2;
    vertices[3 * pId + 2] = v3;

    float3 triN = cross(v2 - v1, v3 - v1);
    triN        = normalize(triN);

    normals[3 * pId + 0] = triN;
    normals[3 * pId + 1] = triN;
    normals[3 * pId + 2] = triN;

    colors[3 * pId + 0] = color;
    colors[3 * pId + 1] = color;
    colors[3 * pId + 2] = color;
}

RigidMeshRender::~RigidMeshRender()
{
    vertices.release();
    normals.release();
    colors.release();
}

bool RigidMeshRender::initializeImpl()
{
    Node* parent = getParent();
    if (parent == NULL)
    {
        Log::sendMessage(Log::Error, "Should insert this module into a node!");
        return false;
    }

    auto triSet = TypeInfo::cast<TriangleSet<DataType3f>>(parent->getTopologyModule());
    if (triSet == nullptr)
    {
        Log::sendMessage(Log::Error, "TriangleModule: The topology module is not supported!");
        return false;
    }

    //		point_render_util = std::make_shared<PointRenderUtil>();

    auto triangles = triSet->getTriangles();

    m_triangleRender = std::make_shared<RigidTriangleRender>();
    m_triangleRender->resize(triangles->size());

    vertices.resize(3 * triangles->size());
    normals.resize(3 * triangles->size());
    colors.resize(3 * triangles->size());

    //updateRenderingContext();
    {
        auto                 verts     = triSet->getPoints();
        auto                 triangles = triSet->getTriangles();
        uint                 pDims     = cudaGridSize(triangles->size(), BLOCK_SIZE);
        DeviceArray<float3>* fverts    = ( DeviceArray<float3>* )&verts;
        SetupRigidTriangles<<<pDims, BLOCK_SIZE>>>(*fverts, vertices, normals, colors, *triangles, make_float3(m_color[0], m_color[1], m_color[2]));

        m_triangleRender->setVertexArray(vertices);
        m_triangleRender->setColorArray(colors);
        m_triangleRender->setNormalArray(normals);
    }
}

void RigidMeshRender::updateRenderingContext()
{
}

void RigidMeshRender::display()
{

    glMatrixMode(GL_MODELVIEW_MATRIX);
    glPushMatrix();

    // Set uniform global transformation.
    Vector3f rotateAxis;
    float    rotateRadian;
    m_rotation.getRotation(rotateRadian, rotateAxis);
    glRotatef(rotateRadian, rotateAxis[0], rotateAxis[1], rotateAxis[2]);
    //glRotatef(m_rotation.x(), m_rotation.y(), m_rotation.z(), m_rotation.w());
    glTranslatef(m_translation[0], m_translation[1], m_translation[2]);
    glScalef(m_scale[0], m_scale[1], m_scale[2]);

    // Update triangle mesh transformation
    m_triangleRender->setRotation(Quaternion<float>(m_meshFrame->getOrientation()));
    m_triangleRender->setTranslatioin(m_meshFrame->getCenter());

    // Display triangle mesh.
    m_triangleRender->display();

    glPopMatrix();
}

void RigidMeshRender::setColor(Vector3f color)
{
    m_color = color;
}

}  // namespace PhysIKA